#include "hip/hip_runtime.h"
#include "mlbm.cuh"
#include "globalStructs.h"
#include "globalFunctions.h"

__global__ void gpuMomCollisionStream(
	dfloat *fMom, unsigned int *dNodeType, ghostInterfaceData ghostInterface, unsigned int step)
{
	const int x = threadIdx.x + blockDim.x * blockIdx.x;
	const int y = threadIdx.y + blockDim.y * blockIdx.y;

	if (x >= NX || y >= NY)
		return;
	dfloat pop[Q];
	__shared__ dfloat s_pop[BLOCK_LBM_SIZE * (Q - 1)];

	// Load moments from global memory

	// rho'
	unsigned int nodeType = dNodeType[idxScalarBlock(threadIdx.x, threadIdx.y, blockIdx.x, blockIdx.y)];
	if (nodeType == 0b11111111)
		return;
	dfloat rhoVar = RHO_0 + fMom[idxMom(threadIdx.x, threadIdx.y, M_RHO_INDEX, blockIdx.x, blockIdx.y)];
	dfloat ux_t30 = fMom[idxMom(threadIdx.x, threadIdx.y, M_UX_INDEX, blockIdx.x, blockIdx.y)];
	dfloat uy_t30 = fMom[idxMom(threadIdx.x, threadIdx.y, M_UY_INDEX, blockIdx.x, blockIdx.y)];
	dfloat m_xx_t45 = fMom[idxMom(threadIdx.x, threadIdx.y, M_MXX_INDEX, blockIdx.x, blockIdx.y)];
	dfloat m_xy_t90 = fMom[idxMom(threadIdx.x, threadIdx.y, M_MXY_INDEX, blockIdx.x, blockIdx.y)];
	dfloat m_yy_t45 = fMom[idxMom(threadIdx.x, threadIdx.y, M_MYY_INDEX, blockIdx.x, blockIdx.y)];

	pop_reconstruction(rhoVar, ux_t30, uy_t30, m_xx_t45, m_yy_t45, m_xy_t90, pop);

	const unsigned short int xp1 = (threadIdx.x + 1 + BLOCK_NX) % BLOCK_NX;
	const unsigned short int xm1 = (threadIdx.x - 1 + BLOCK_NX) % BLOCK_NX;

	const unsigned short int yp1 = (threadIdx.y + 1 + BLOCK_NY) % BLOCK_NY;
	const unsigned short int ym1 = (threadIdx.y - 1 + BLOCK_NY) % BLOCK_NY;

	// save populations in shared memory
	s_pop[idxPopBlock(threadIdx.x, threadIdx.y, 0)] = pop[1];
	s_pop[idxPopBlock(threadIdx.x, threadIdx.y, 1)] = pop[2];
	s_pop[idxPopBlock(threadIdx.x, threadIdx.y, 2)] = pop[3];
	s_pop[idxPopBlock(threadIdx.x, threadIdx.y, 3)] = pop[4];
	s_pop[idxPopBlock(threadIdx.x, threadIdx.y, 4)] = pop[5];
	s_pop[idxPopBlock(threadIdx.x, threadIdx.y, 5)] = pop[6];
	s_pop[idxPopBlock(threadIdx.x, threadIdx.y, 6)] = pop[7];
	s_pop[idxPopBlock(threadIdx.x, threadIdx.y, 7)] = pop[8];

	// sync threads of the block so all populations are saved
	__syncthreads();

	/* pull */

	pop[1] = s_pop[idxPopBlock(xm1, threadIdx.y, 0)];
	pop[2] = s_pop[idxPopBlock(threadIdx.x, ym1, 1)];
	pop[3] = s_pop[idxPopBlock(xp1, threadIdx.y, 2)];
	pop[4] = s_pop[idxPopBlock(threadIdx.x, yp1, 3)];
	pop[5] = s_pop[idxPopBlock(xm1, ym1, 4)];
	pop[6] = s_pop[idxPopBlock(xp1, ym1, 5)];
	pop[7] = s_pop[idxPopBlock(xp1, yp1, 6)];
	pop[8] = s_pop[idxPopBlock(xm1, yp1, 7)];

	const int tx = threadIdx.x;
	const int ty = threadIdx.y;

	const int bx = blockIdx.x;
	const int by = blockIdx.y;

	const int txm1 = (tx - 1 + BLOCK_NX) % BLOCK_NX;
	const int txp1 = (tx + 1 + BLOCK_NX) % BLOCK_NX;

	const int tym1 = (ty - 1 + BLOCK_NY) % BLOCK_NY;
	const int typ1 = (ty + 1 + BLOCK_NY) % BLOCK_NY;

	const int bxm1 = (bx - 1 + NUM_BLOCK_X) % NUM_BLOCK_X;
	const int bxp1 = (bx + 1 + NUM_BLOCK_X) % NUM_BLOCK_X;

	const int bym1 = (by - 1 + NUM_BLOCK_Y) % NUM_BLOCK_Y;
	const int byp1 = (by + 1 + NUM_BLOCK_Y) % NUM_BLOCK_Y;

	/* load pop from global in cover nodes */

#include "includeFiles/popLoad.inc"

	dfloat invRho;

	if (nodeType != BULK)
	{
#ifndef CYLINDER
		boundary_calculation(nodeType, &rhoVar, &ux_t30, &uy_t30, &m_xx_t45, &m_yy_t45, &m_xy_t90, pop, fMom, x, y);
#endif
		invRho = 1.0 / rhoVar;
	}
	else
	{
		rhoVar = pop[0] + pop[1] + pop[2] + pop[3] + pop[4] + pop[5] + pop[6] + pop[7] + pop[8];
		invRho = 1 / rhoVar;

		ux_t30 = ((pop[1] + pop[5] + pop[8]) - (pop[3] + pop[6] + pop[7])) * invRho;
		uy_t30 = ((pop[2] + pop[5] + pop[6]) - (pop[4] + pop[7] + pop[8])) * invRho;

		m_xx_t45 = (pop[1] + pop[3] + pop[5] + pop[6] + pop[7] + pop[8]) * invRho - cs2;
		m_xy_t90 = ((pop[5] + pop[7]) - (pop[6] + pop[8])) * invRho;
		m_yy_t45 = (pop[2] + pop[4] + pop[5] + pop[6] + pop[7] + pop[8]) * invRho - cs2;
	}

	ux_t30 = F_M_I_SCALE * ux_t30;
	uy_t30 = F_M_I_SCALE * uy_t30;

	m_xx_t45 = F_M_II_SCALE * (m_xx_t45);
	m_xy_t90 = F_M_IJ_SCALE * (m_xy_t90);
	m_yy_t45 = F_M_II_SCALE * (m_yy_t45);

// COLLIDE
#ifndef CYLINDER
	moment_collision(ux_t30, uy_t30, &m_xx_t45, &m_yy_t45, &m_xy_t90);
#endif

	// calculate post collision populations
	pop_reconstruction(rhoVar, ux_t30, uy_t30, m_xx_t45, m_yy_t45, m_xy_t90, pop);

	/* write to global mom */

	fMom[idxMom(threadIdx.x, threadIdx.y, M_RHO_INDEX, blockIdx.x, blockIdx.y)] = rhoVar - RHO_0;

	fMom[idxMom(threadIdx.x, threadIdx.y, M_UX_INDEX, blockIdx.x, blockIdx.y)] = ux_t30;
	fMom[idxMom(threadIdx.x, threadIdx.y, M_UY_INDEX, blockIdx.x, blockIdx.y)] = uy_t30;

	fMom[idxMom(threadIdx.x, threadIdx.y, M_MXX_INDEX, blockIdx.x, blockIdx.y)] = m_xx_t45;
	fMom[idxMom(threadIdx.x, threadIdx.y, M_MXY_INDEX, blockIdx.x, blockIdx.y)] = m_xy_t90;
	fMom[idxMom(threadIdx.x, threadIdx.y, M_MYY_INDEX, blockIdx.x, blockIdx.y)] = m_yy_t45;

#include "includeFiles/popSave.inc"
}
